
#include <hip/hip_runtime.h>
/*
  -- MAGMA (version 1.1) --
  Univ. of Tennessee, Knoxville
  Univ. of California, Berkeley
  Univ. of Colorado, Denver
  November 2011


  @precisions normal z -> z c d s
       
*/

#if (CUDA_SM_VERSION == 20) || (CUDA_SM_VERSION == 30)

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <cuda.h>
#include <cublas.h>

#include "dague.h"
#include "data_dist/matrix/precision.h"

#define PRECISION_z

#if defined(PRECISION_z) || defined(PRECISION_c) 
#include <cuComplex.h>
#endif

///////////////////////////////////////////////////////////////////////////////////////////////////

#define GENERATE_SM_VERSION_KERNEL_NAME_I(func, version)  zgemm_##func##_SM##version
#define GENERATE_SM_VERSION_KERNEL_NAME_I2(func, version) GENERATE_SM_VERSION_KERNEL_NAME_I(func, version)
#define GENERATE_SM_VERSION_KERNEL_NAME(func)             GENERATE_SM_VERSION_KERNEL_NAME_I2(func, CUDA_SM_VERSION)

#define GENERATE_SM_VERSION_NAME_I(func, version)  magmablas_##func##_SM##version
#define GENERATE_SM_VERSION_NAME_I2(func, version) GENERATE_SM_VERSION_NAME_I(func, version)
#define GENERATE_SM_VERSION_NAME(func)             GENERATE_SM_VERSION_NAME_I2(func, CUDA_SM_VERSION)

///////////////////////////////////////////////////////////////////////////////////////////////////
#if 1
extern "C" void
GENERATE_SM_VERSION_NAME(zgemm)( char TRANSA, char TRANSB, int m, int n, int k,
                                 dague_complex64_t alpha, dague_complex64_t *d_A, int lda,
                                                          dague_complex64_t *d_B, int ldb,
                                 dague_complex64_t beta,  dague_complex64_t *d_C, int ldc,
                                 CUstream stream )
{
#if defined(PRECISION_z) || defined(PRECISION_c)    
    cuDoubleComplex lalpha = make_cuDoubleComplex( creal(alpha), cimag(alpha) );
    cuDoubleComplex lbeta  = make_cuDoubleComplex( creal(beta),  cimag(beta)  );
#else
    double lalpha = alpha;
    double lbeta  = beta;
#endif

#if (__CUDA_API_VERSION < 4000)
    
    cublasSetKernelStream( stream );

    cublasZgemm(TRANSA, TRANSB, m, n, k, 
                lalpha, (cuDoubleComplex*)d_A, lda,
                        (cuDoubleComplex*)d_B, ldb,
                lbeta,  (cuDoubleComplex*)d_C, ldc); 

#else
    cudaStream_t current_stream;
    cublasHandle_t handle = cublasGetCurrentCtx();
    
    cublasGetStream_v2 ( handle, &saved_stream );
    cublasSetStream_v2 ( handle, &stream );

    cublasZgemm_v2(handle, convertToOp(TRANSA), convertToOp(TRANSB),
                   m, n, k, 
                   &lalpha, (cuDoubleComplex*)d_A, lda,
                            (cuDoubleComplex*)d_B, ldb,
                   &lbeta,  (cuDoubleComplex*)d_C, ldc); 

    cublasSetStream_v2 ( handle, &saved_stream );
#endif
}

#else
#include "gemm_stencil_20_30.h"

/*  -- MAGMA (version 1.1) --
    Univ. of Tennessee, Knoxville
    Univ. of California, Berkeley
    Univ. of Colorado, Denver
    November 2011

    Purpose
    =======
    ZGEMM  performs one of the matrix-matrix operations

    C := alpha*op( A )*op( B ) + beta*C,

    where  op( X ) is one of

    op( X ) = X   or   op( X ) = X',

    alpha and beta are scalars, and A, B and C are matrices, with op( A )
    an m by k matrix,  op( B )  a  k by n matrix and  C an m by n matrix.

    Parameters
    ==========
    TRANSA - CHARACTER*1.
    On entry, TRANSA specifies the form of op( A ) to be used in
    the matrix multiplication as follows:
    TRANSA = 'N' or 'n',  op( A ) = A.
    TRANSA = 'T' or 't',  op( A ) = A'.
    TRANSA = 'C' or 'c',  op( A ) = A'.
    Unchanged on exit.

    TRANSB - CHARACTER*1.
    On entry, TRANSB specifies the form of op( B ) to be used in
    the matrix multiplication as follows:
    TRANSB = 'N' or 'n',  op( B ) = B.
    TRANSB = 'T' or 't',  op( B ) = B'.
    TRANSB = 'C' or 'c',  op( B ) = B'.
    Unchanged on exit.

    M      - INTEGER.
    On entry,  M  specifies  the number  of rows  of the  matrix
    op( d_A )  and of the  matrix d_C.  M  must  be at least  zero.
    Unchanged on exit.

    N      - INTEGER.
    On entry,  N  specifies the number  of columns of the matrix
    op( d_B ) and the number of columns of the matrix d_C. N must be
    at least zero.
    Unchanged on exit.

    K      - INTEGER.
    On entry,  K  specifies  the number of columns of the matrix
    op( d_A ) and the number of rows of the matrix op( d_B ). K must
    be at least  zero.
    Unchanged on exit.

    ALPHA  - COMPLEX_16
    On entry, ALPHA specifies the scalar alpha.
    Unchanged on exit.

    d_A    - COMPLEX_16 array of DIMENSION ( LDA, ka ), where ka is
    k  when  TRANSA = 'N' or 'n',  and is  m  otherwise.
    Before entry with  TRANSA = 'N' or 'n',  the leading  m by k
    part of the array d_A must contain the matrix d_A, otherwise
    the leading  k by m  part of the array d_A must contain  the
    matrix d_A.
    Unchanged on exit.

    LDA    - INTEGER.
    On entry, LDA specifies the first dimension of A as declared
    in the calling (sub) program. When  TRANSA = 'N' or 'n' then
    LDA must be at least  max( 1, m ), otherwise  LDA must be at
    least  max( 1, k ).
    Unchanged on exit.

    d_B    - COMPLEX_16 array of DIMENSION ( LDB, kb ), where kb is
    n  when  TRANSB = 'N' or 'n',  and is  k  otherwise.
    Before entry with  TRANSB = 'N' or 'n',  the leading  k by n
    part of the array d_B must contain the matrix d_B, otherwise
    the leading  n by k  part of the array d_B must contain  the
    matrix d_B.
    Unchanged on exit.
 
    LDB    - INTEGER.
    On entry, LDB specifies the first dimension of d_B as declared
    in the calling (sub) program. When  TRANSB = 'N' or 'n' then
    LDB must be at least  max( 1, k ), otherwise  LDB must be at
    least  max( 1, n ).
    Unchanged on exit.

    BETA   - COMPLEX_16.
    On entry,  BETA  specifies the scalar  beta.  When  BETA  is
    supplied as zero then d_C need not be set on input.
    Unchanged on exit.

    d_C    - COMPLEX_16 array of DIMENSION ( LDC, n ).
    Before entry, the leading  m by n  part of the array  d_C must
    contain the matrix  d_C,  except when  beta  is zero, in which
    case d_C need not be set on entry.
    On exit, the array  d_C  is overwritten by the  m by n  matrix
    ( alpha*op( d_A )*op( d_B ) + beta*d_C ).

    LDC    - INTEGER.
    On entry, LDC specifies the first dimension of d_C as declared
    in  the  calling  (sub)  program.   LDC  must  be  at  least
    max( 1, m ).
    Unchanged on exit.
    =====================================================================    */

extern "C" void
GENERATE_SM_VERSION_NAME(zgemm)( char TRANSA, char TRANSB, int m, int n, int k,
                                 dague_complex64_t alpha, dague_complex64_t *d_A, int lda,
                                                          dague_complex64_t *d_B, int ldb,
                                 dague_complex64_t beta,  dague_complex64_t *d_C, int ldc,
                                 CUstream stream )
{
    if (m<=0 || n<=0 || k<=0)
        return;

#if defined(PRECISION_z) || defined(PRECISION_c)    
    cuDoubleComplex lalpha = make_cuDoubleComplex( creal(alpha), cimag(alpha) );
    cuDoubleComplex lbeta  = make_cuDoubleComplex( creal(beta),  cimag(beta)  );
#else
    double lalpha = alpha;
    double lbeta  = beta;
#endif

    size_t offsetA = 0;
    size_t offsetB = 0;

#if defined(PRECISION_z) || defined(PRECISION_c)
    int TransA = 2, TransB = 2;
#else
    int TransA = 1, TransB = 1;
#endif

    if (TRANSA == 'T' ||  TRANSA == 't')
        TransA = 1;
    else
        if (TRANSA == 'N' ||  TRANSA == 'n')
            TransA = 0;
    
    if (TRANSB == 'T' ||  TRANSB == 't')
        TransB = 1;
    else
        if (TRANSB == 'N' ||  TRANSB == 'n')
            TransB = 0;

#ifdef TEXTURE_1D
    size_t sizeA = (size_t) lda * (size_t) (!TransA ? k : m);
    size_t sizeB = (size_t) ldb * (size_t) (!TransB ? n : k);

    /* TODO: Check with Jakub what is this */
    size_t CUBLAS_MAX_1DBUF_SIZE = ((1 << 27) - 512);
#if 0
    if (sizeA>=CUBLAS_MAX_1DBUF_SIZE ||
        sizeB>=CUBLAS_MAX_1DBUF_SIZE )
        {
            cublasZgemm(TRANSA, TRANSB, m, n, k, 
                        lalpha, (cuDoubleComplex*)d_A, lda, 
                               (cuDoubleComplex*)d_B, ldb,
                        lbeta,  (cuDoubleComplex*)d_C, ldc);
            return;
        }
#else
    if (sizeA>=CUBLAS_MAX_1DBUF_SIZE ||
        sizeB>=CUBLAS_MAX_1DBUF_SIZE )
        {
            fprintf(stderr, "ERROR: The matrix size is too big to use texture\n");
            return;
        }

#endif

    // Set textures parameters
    tex_ref_A.normalized = false;
    tex_ref_A.filterMode = cudaFilterModePoint;
    tex_ref_A.addressMode[0] = cudaAddressModeClamp;
    
    tex_ref_B.normalized = false;
    tex_ref_B.filterMode = cudaFilterModePoint;
    tex_ref_B.addressMode[0] = cudaAddressModeClamp;
    
    // Bind A and B to texture references
    assert(cudaBindTexture(&offsetA, tex_ref_A, d_A, sizeA*sizeof(dague_complex64_t)) 
           == cudaSuccess);
    assert(cudaBindTexture(&offsetB, tex_ref_B, d_B, sizeB*sizeof(dague_complex64_t))
           == cudaSuccess);
#endif

    // Set up grids
    // Warning: works because DIM_X and DIM_Y are equals for every cases of one precision
    dim3 dimBlock(DIM_X, DIM_Y);

    offsetA = offsetA/sizeof(d_A[0]);
    offsetB = offsetB/sizeof(d_B[0]);
    
    if (TransA==0 && TransB ==0){
        dim3 dimGrid(m/BLK_M_nn + (m%BLK_M_nn != 0),
                     n/BLK_N_nn + (n%BLK_N_nn != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(nn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, 
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    } 
    else if (TransA==0 && TransB ==1){
        dim3 dimGrid(m/BLK_M_nt + (m%BLK_M_nt != 0),
                     n/BLK_N_nt + (n%BLK_N_nt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(nt)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
    else if (TransA==1 && TransB ==0){
        dim3 dimGrid(m/BLK_M_tn + (m%BLK_M_tn != 0),
                     n/BLK_N_tn + (n%BLK_N_tn != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(tn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
    else if (TransA==1 && TransB ==1){
        dim3 dimGrid(m/BLK_M_tt + (m%BLK_M_tt != 0),
                     n/BLK_N_tt + (n%BLK_N_tt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(tt)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
#if defined(PRECISION_z) || defined(PRECISION_c) 
    else if (TransA==0 && TransB ==2){
        dim3 dimGrid(m/BLK_M_nt + (m%BLK_M_nt != 0),
                     n/BLK_N_nt + (n%BLK_N_nt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(nc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    } 
    else if (TransA==1 && TransB ==2){
        dim3 dimGrid(m/BLK_M_tt + (m%BLK_M_tt != 0),
                     n/BLK_N_tt + (n%BLK_N_tt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(tc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
    else if (TransA==2 && TransB ==0){
        dim3 dimGrid(m/BLK_M_tn + (m%BLK_M_tn != 0),
                     n/BLK_N_tn + (n%BLK_N_tn != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(cn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
    else if (TransA==2 && TransB ==1){
        dim3 dimGrid(m/BLK_M_tt + (m%BLK_M_tt != 0),
                     n/BLK_N_tt + (n%BLK_N_tt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(ct)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    } 
    else if (TransA==2 && TransB ==2){
        dim3 dimGrid(m/BLK_M_tt + (m%BLK_M_tt != 0),
                     n/BLK_N_tt + (n%BLK_N_tt != 0));
        GENERATE_SM_VERSION_KERNEL_NAME(cc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k,
                                                                                lalpha, (cuDoubleComplex*)d_A, lda,
                                                                                        (cuDoubleComplex*)d_B, ldb,
                                                                                lbeta,  (cuDoubleComplex*)d_C, ldc,
                                                                                (int)offsetA, (int)offsetB);
    }
#endif

#ifdef TEXTURE_1D
    cudaUnbindTexture ( tex_ref_A ) ;
    cudaUnbindTexture ( tex_ref_B ) ;
#endif
}

#endif

///////////////////////////////////////////////////////////////////////////////////////////////////
#endif /* (CUDA_SM_VERSION == 20) || (CUDA_SM_VERSION == 30) */

