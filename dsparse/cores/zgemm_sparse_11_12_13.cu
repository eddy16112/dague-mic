
#include <hip/hip_runtime.h>
/*
  -- MAGMA (version 1.1) --
  Univ. of Tennessee, Knoxville
  Univ. of California, Berkeley
  Univ. of Colorado, Denver
  November 2011


  @precisions normal z -> z c d s
       
*/

#if (CUDA_SM_VERSION == 11) || (CUDA_SM_VERSION == 12) || (CUDA_SM_VERSION == 13)

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <cuda.h>
#include <cublas.h>

#include "dague.h"
#include "data_dist/matrix/precision.h"

#define PRECISION_z

#if defined(PRECISION_z) || defined(PRECISION_c) 
#include <cuComplex.h>
#endif

///////////////////////////////////////////////////////////////////////////////////////////////////

#define GENERATE_SM_VERSION_KERNEL_NAME_I(func, version)  kernel_zgemm_##func##_SM##version
#define GENERATE_SM_VERSION_KERNEL_NAME_I2(func, version) GENERATE_SM_VERSION_KERNEL_NAME_I(func, version)
#define GENERATE_SM_VERSION_KERNEL_NAME(func)             GENERATE_SM_VERSION_KERNEL_NAME_I2(func, CUDA_SM_VERSION)

#define GENERATE_SM_VERSION_NAME_I(func, version)  func##_SM##version
#define GENERATE_SM_VERSION_NAME_I2(func, version) GENERATE_SM_VERSION_NAME_I(func, version)
#define GENERATE_SM_VERSION_NAME(func)             GENERATE_SM_VERSION_NAME_I2(func, CUDA_SM_VERSION)

///////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ void kernel_zaxpy(const cuDoubleComplex a, 
                                    const cuDoubleComplex *b,
                                          cuDoubleComplex *c)
{
    c[0]  += a * b[0];
    c[1]  += a * b[1];
    c[2]  += a * b[2];
    c[3]  += a * b[3];
    c[4]  += a * b[4];
    c[5]  += a * b[5];
    c[6]  += a * b[6];
    c[7]  += a * b[7];
    c[8]  += a * b[8];
    c[9]  += a * b[9];
    c[10] += a * b[10];
    c[11] += a * b[11];
    c[12] += a * b[12];
    c[13] += a * b[13];
    c[14] += a * b[14];
    c[15] += a * b[15];
}

extern "C" __global__ void
GENERATE_SM_VERSION_KERNEL_NAME(nt)(int m, int n, int k,
                                    cuDoubleComplex alpha, const cuDoubleComplex *A, int lda,
                                                           const cuDoubleComplex *B, int ldb, 
                                    cuDoubleComplex beta,        cuDoubleComplex *C, int ldc,
                                    int blocknbr, const int *blocktab, int fblocknbr, const int *fblocktab)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int ibx = blockIdx.x * 64;
    const int iby = blockIdx.y * 16;
        
    const int idt = ty * 16 + tx;

    if( iby + tx >=n )
        B += iby+0;
    else
        B += iby+tx;
    /*
      Taking care of boundary cases where K<4.
    */
    if( ty >=k ) 
        B+= __mul24( 0, ldb );
    else
        B+= __mul24( ty,ldb );
                
    if( ibx + idt >= m ) 
        A += ibx + 0; 
    else
        A += ibx + idt;

    int s2=lda, s3=2*lda, s4=3*lda; 

    switch (k){
    case 1: 
        s2=0; s3=0; s4=0; 
        break ; 
    case 2:
        s2=lda; s3=0; s4=0; 
        break ; 
    case 3:  
        s2=lda; s3=2*lda; s4=0; 
        break ;
    }

    if (ibx + idt < m)
    {
#define FROWNUM(tab, b) tab[2*b]
#define LROWNUM(tab, b) tab[2*b+1]
#define BLOCKSIZE(tab, b) LROWNUM(tab, b) - FROWNUM(tab, b) + 1
        int idx_x = ibx + idt;
        int blocknum = 0, fblocknum = 0;
        size_t totalblocksize = 0;
        size_t blocksize = BLOCKSIZE(blocktab, blocknum);
        int rownum;
        int offset;

        /*
         * We should keep blocknum < blocknbr
         */
        while(totalblocksize + blocksize < idx_x + 1) 
            {
                totalblocksize += blocksize;
                blocknum++;
                blocksize = BLOCKSIZE(blocktab, blocknum);
            }
        rownum = idx_x - totalblocksize + FROWNUM(blocktab, blocknum);
        offset = 0;
        while (LROWNUM(fblocktab, fblocknum) < rownum) {
            offset += BLOCKSIZE(fblocktab, fblocknum);
            fblocknum++;
        }
        offset += rownum - FROWNUM(fblocktab, fblocknum);
        
        C += offset + __mul24(iby,ldc);
#undef FROWNUM
#undef LROWNUM
    } 
    //__syncthreads();

    cuDoubleComplex Ap[4] = {A[0], A[s2], A[s3], A[s4]};
    cuDoubleComplex b = B[0];

    const cuDoubleComplex *Bend = B + ldb*(k-k%4);

    B+=4*ldb;
    A+=4*lda;

    __shared__ cuDoubleComplex Bb[4][16];

    cuDoubleComplex Cb[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    if(k>7)
        do {
            cuDoubleComplex Ab[4] = {Ap[0], Ap[1], Ap[2], Ap[3]};

            Bb[ty][tx]=b;

            __syncthreads();

            Ap[0] = A[0];
            Ap[1] = A[s2];
            Ap[2] = A[s3];
            Ap[3] = A[s4];

            b=B[0];

            kernel_zaxpy(Ab[0], &Bb[0][0], Cb);
            kernel_zaxpy(Ab[1], &Bb[1][0], Cb);
            kernel_zaxpy(Ab[2], &Bb[2][0], Cb);
            kernel_zaxpy(Ab[3], &Bb[3][0], Cb);

            A += 4*lda;
            B += 4*ldb;

            __syncthreads();
        } while (B < Bend);

    if(k>3){

        Bb[ty][tx]=b;
        int k1 = k-k%4;

        if( (k1+ty) >=k)
            B-=4*ldb;
        else 
            B-=0*ldb;

        if( (k1+0) >= k ) {s2=0; s3=0*lda; s4=0; A-=4*lda;} else
            if( (k1+1) >= k ) {s2=0; s3=0*lda; s4=0; A-=0*lda;} else
                if( (k1+2) >= k ) {s2=lda; s3=0*lda; s4=0; A-=0*lda;} else
                    if( (k1+3) >= k ) {s2=lda; s3=2*lda; s4=0; A-=0*lda;} 
                        
        __syncthreads();

        b=B[0];

        kernel_zaxpy(Ap[0], &Bb[0][0], Cb);        Ap[0] = A[0];
        kernel_zaxpy(Ap[1], &Bb[1][0], Cb);        Ap[1] = A[s2];
        kernel_zaxpy(Ap[2], &Bb[2][0], Cb);        Ap[2] = A[s3];
        kernel_zaxpy(Ap[3], &Bb[3][0], Cb);        Ap[3] = A[s4];
        
    }

    k=k%4;

    if ( k!=0){

        __syncthreads();

        Bb[ty][tx]=b;

        __syncthreads();

        for(int i=0;i<k;i++){
            kernel_zaxpy(Ap[i],&Bb[i][0], Cb);
        }
    }

    if( (iby+16)>=n) { 
        lda = n-iby;
    }
    else{
        lda = 16;
    }

    if( (ibx+idt) >= m )
        lda = 0 ;
    else lda = lda ;

    switch(lda){
    case 16:
        
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        C[11*ldc] = alpha*Cb[11] + beta * C[11*ldc];
        C[12*ldc] = alpha*Cb[12] + beta * C[12*ldc];
        C[13*ldc] = alpha*Cb[13] + beta * C[13*ldc];
        C[14*ldc] = alpha*Cb[14] + beta * C[14*ldc];
        C[15*ldc] = alpha*Cb[15] + beta * C[15*ldc];

        break;
    case 15:
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        C[11*ldc] = alpha*Cb[11] + beta * C[11*ldc];
        C[12*ldc] = alpha*Cb[12] + beta * C[12*ldc];
        C[13*ldc] = alpha*Cb[13] + beta * C[13*ldc];
        C[14*ldc] = alpha*Cb[14] + beta * C[14*ldc];
        break;
    case 14:
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        C[11*ldc] = alpha*Cb[11] + beta * C[11*ldc];
        C[12*ldc] = alpha*Cb[12] + beta * C[12*ldc];
        C[13*ldc] = alpha*Cb[13] + beta * C[13*ldc];
        break;
    case 13:
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        C[11*ldc] = alpha*Cb[11] + beta * C[11*ldc];
        C[12*ldc] = alpha*Cb[12] + beta * C[12*ldc];
        break;
    case 12:
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        C[11*ldc] = alpha*Cb[11] + beta * C[11*ldc];
        break;
    case 11:
        C[0]      = alpha*Cb[0]  + beta * C[0];
        C[1*ldc]  = alpha*Cb[1]  + beta * C[1*ldc];
        C[2*ldc]  = alpha*Cb[2]  + beta * C[2*ldc];
        C[3*ldc]  = alpha*Cb[3]  + beta * C[3*ldc];
        C[4*ldc]  = alpha*Cb[4]  + beta * C[4*ldc];
        C[5*ldc]  = alpha*Cb[5]  + beta * C[5*ldc];
        C[6*ldc]  = alpha*Cb[6]  + beta * C[6*ldc];
        C[7*ldc]  = alpha*Cb[7]  + beta * C[7*ldc];
        C[8*ldc]  = alpha*Cb[8]  + beta * C[8*ldc];
        C[9*ldc]  = alpha*Cb[9]  + beta * C[9*ldc];
        C[10*ldc] = alpha*Cb[10] + beta * C[10*ldc];
        break;
    case 10:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        C[5*ldc] = alpha*Cb[5] + beta * C[5*ldc];
        C[6*ldc] = alpha*Cb[6] + beta * C[6*ldc];
        C[7*ldc] = alpha*Cb[7] + beta * C[7*ldc];
        C[8*ldc] = alpha*Cb[8] + beta * C[8*ldc];
        C[9*ldc] = alpha*Cb[9] + beta * C[9*ldc];
        break;
    case 9:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        C[5*ldc] = alpha*Cb[5] + beta * C[5*ldc];
        C[6*ldc] = alpha*Cb[6] + beta * C[6*ldc];
        C[7*ldc] = alpha*Cb[7] + beta * C[7*ldc];
        C[8*ldc] = alpha*Cb[8] + beta * C[8*ldc];
        break;
    case 8:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        C[5*ldc] = alpha*Cb[5] + beta * C[5*ldc];
        C[6*ldc] = alpha*Cb[6] + beta * C[6*ldc];
        C[7*ldc] = alpha*Cb[7] + beta * C[7*ldc];
        break;
    case 7:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        C[5*ldc] = alpha*Cb[5] + beta * C[5*ldc];
        C[6*ldc] = alpha*Cb[6] + beta * C[6*ldc];
        break;
    case 6:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        C[5*ldc] = alpha*Cb[5] + beta * C[5*ldc];
        break;
    case 5:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        C[4*ldc] = alpha*Cb[4] + beta * C[4*ldc];
        break;
    case 4:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        C[3*ldc] = alpha*Cb[3] + beta * C[3*ldc];
        break;
    case 3:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        C[2*ldc] = alpha*Cb[2] + beta * C[2*ldc];
        break;
    case 2:
        C[0]     = alpha*Cb[0] + beta * C[0];
        C[1*ldc] = alpha*Cb[1] + beta * C[1*ldc];
        break;
    case 1:
        C[0] = alpha*Cb[0] + beta * C[0];
        break;
    case 0:
        break;
    }

}

extern "C" void
GENERATE_SM_VERSION_NAME(zgemm_sparse)( char TRANSA, char TRANSB, int m, int n, int k,
                                        dague_complex64_t alpha, dague_complex64_t *d_A, int lda,
                                                                 dague_complex64_t *d_B, int ldb,
                                        dague_complex64_t beta,  dague_complex64_t *d_C, int ldc,
                                        int blocknbr, const int *blocktab, int fblocknbr, const int *fblocktab,
                                        CUstream stream )
{
#if defined(PRECISION_z) || defined(PRECISION_c)    
    cuDoubleComplex lalpha = make_cuDoubleComplex( creal(alpha), cimag(alpha) );
    cuDoubleComplex lbeta  = make_cuDoubleComplex( creal(beta),  cimag(beta)  );
#else
    double lalpha = alpha;
    double lbeta  = beta;
#endif

#if defined(PRECISION_z)
    dim3 threads( 16, 4 );
    dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
#else
    dim3 threads( 16, 4 );
    dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
#endif
    GENERATE_SM_VERSION_KERNEL_NAME(nt)
        <<< grid, threads, 0, stream >>>(m, n, k,
                                         lalpha, (cuDoubleComplex*)d_A, lda,
                                                 (cuDoubleComplex*)d_B, ldb,
                                         lbeta,  (cuDoubleComplex*)d_C, ldc,
                                         blocknbr, blocktab, fblocknbr, fblocktab);
}

///////////////////////////////////////////////////////////////////////////////////////////////////
#endif /* (CUDA_SM_VERSION == 11) || (CUDA_SM_VERSION == 12) || (CUDA_SM_VERSION == 13) */

